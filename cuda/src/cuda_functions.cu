#include "hip/hip_runtime.h"
#include <cuda_functions.h>


using namespace std;

__global__ void calculate(int * array_food){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    array_food[idx] = array_food[idx] + 1;
}


// 0 Position and type obj
// 1 age
// 2 safety points
// 3 grow points
// 4 repr points
// 5 health points


__device__ int direction[8][2] = {{-1, -1}, {0, -1}, {1, -1},
                                  {-1, 0}, {1, 0},
                                  {-1, 1}, {0, 1}, {1, 1}};


__device__ void check_safety(int array_food[][W][H], int idx, int idy, int c_max, bool &safe){
    int c = 0;
    safe = false;
    for (auto route : direction){
        int pos_x = idx + route[0];
        int pos_y = idy + route[1];
        if(pos_x >= 0 && pos_y >= 0 && pos_x < W && pos_y < H){
            if (array_food[0][pos_x][pos_y] == 1 && array_food[5][pos_x][pos_y] >= 3){
                c++;
            }
        } else {
            c++;
        }
    }
    if (c >= c_max){
        safe = true;
    }
}


dim3 dimBlock(W, H);
dim3 dimGrid(1, 1);



__global__ void calculate_obj(int array_food[C][W][H]){
//__global__ void calculate_obj(int ***gpu_vector){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;

    bool safe = false;

    if (array_food[0][idx][idy] == 1){
        /// Remove
        if(array_food[1][idx][idy] == 50){//array_food[5][idx][idy] == 0){
            array_food[0][idx][idy] = 0;
            array_food[1][idx][idy] = 0;
            array_food[2][idx][idy] = 0;
            array_food[3][idx][idy] = 0;
            array_food[4][idx][idy] = 0;
        } else {
            array_food[1][idx][idy] += 1;
            check_safety(array_food, idx, idy, 8, safe);
            /// Check non blocked
            if (safe == false){


                /// Check safety
                check_safety(array_food, idx, idy, 6, safe);

                /// Increase health and size
                if (array_food[1][idx][idy] % 5 == 0){
//                    array_food[1][idx][idy] = 1;  /// ToDo remove
                    if (array_food[5][idx][idy] < 6){
                        array_food[5][idx][idy] += 1;
                    }
                }

                /// Grow inc/dec
                if (array_food[5][idx][idy] >= 3){
                    array_food[3][idx][idy] += 1;
                } else {
                    array_food[3][idx][idy] = 0;
                }

                ///Grow direction
                if (array_food[3][idx][idy] == 3){
                    array_food[3][idx][idy] = 0;
    //                random_shuffle(direction, direction+9);

                    for (auto route : direction){
                        int pos_x = idx + route[0];
                        int pos_y = idy + route[1];

                        if(pos_x >= 0 && pos_y >= 0 && pos_x <= W && pos_y <= H){
                            if (array_food[0][pos_x][pos_y] == 0){
                                /// Create
                                array_food[0][pos_x][pos_y] = 1;
                                array_food[1][pos_x][pos_y] = 1;
                                array_food[5][pos_x][pos_y] = 1;
                            }
                        }
                    }
                }
            } else {
                array_food[5][idx][idy] = 6;
            }
        }
    }
}

CudaFunctions::CudaFunctions(){
    int count;
    hipGetDeviceCount(&count);
    printf("Device: %d\n", count);
    hipDeviceProp_t dev_prop;
    for(int i = 0; i < count; i++){
        hipGetDeviceProperties(&dev_prop, i);
        printf("ID: %d Name: %s\n", i, dev_prop.name);
    }

    int dev;
    memset(&dev_prop, 0, sizeof(hipDeviceProp_t));

    dev_prop.major = 1;
    dev_prop.minor = 3;

    hipGetDevice(&dev);
    printf("ID current: %d\n", dev);
    hipChooseDevice(&dev, &dev_prop);
    printf("ID nearest: %d\n\n", dev);
    hipSetDevice(dev);
}


void CudaFunctions::test(){
    cout << "test" << endl;
}

#include <chrono>

void CudaFunctions::calculate_food(array2D *array_food){
    array2D *gpu_vector;

    hipMalloc(&gpu_vector, C*W*H*C*sizeof(int));
    hipMemcpy(gpu_vector, array_food, C*W*H*sizeof(int), hipMemcpyHostToDevice);

    calculate_obj<<<dimBlock, dimGrid>>>(gpu_vector);
    hipDeviceSynchronize();

    hipMemcpy(array_food, gpu_vector, C*W*H*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpu_vector);
    hipDeviceSynchronize();
}
